#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "raytrace.h"

// CUDA kernel to add two arrays element by element
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    C[i] = A[i] + B[i];
  }
}

// Main program
int main(void) {
  // Print CUDA device information
  int deviceCount = 0;
  float milliseconds = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);

  if (error != hipSuccess) {
    printf("Error: Failed to get CUDA device count: %s\n",
           hipGetErrorString(error));
    return EXIT_FAILURE;
  }

  if (deviceCount == 0) {
    printf("Warning: No CUDA devices found\n");
    return EXIT_SUCCESS;
  }

  printf("Detected %d CUDA device(s)\n", deviceCount);

  // Use first device
  hipSetDevice(0);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  printf("Using device 0: %s\n", deviceProp.name);

  // Set vector size
  int numElements = 50000;
  size_t size = numElements * sizeof(float);
  printf("Vector size: %d\n", numElements);

  // Allocate host memory
  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);

  // Initialize host arrays
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate device memory
  float *d_A = NULL;
  float *d_B = NULL;
  float *d_C = NULL;

  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Launch CUDA kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);

  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("Error: Failed to launch CUDA kernel: %s\n",
           hipGetErrorString(error));
    goto cleanup;
  }

  // Copy result back to host
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Verify result (just checking a few elements)
  for (int i = 0; i < 5; ++i) {
    printf("%.2f + %.2f = %.2f\n", h_A[i], h_B[i], h_C[i]);
  }

  // Measure performance
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Warm up
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

  // Timing run
  hipEventRecord(start);
  for (int i = 0; i < 100; i++) {
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Performance: %f ms per kernel launch (average of 100 launches)\n",
         milliseconds / 100);

cleanup:
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  // Reset device
  hipDeviceReset();

  printf("CUDA demo completed successfully\n");
  return 0;
}
