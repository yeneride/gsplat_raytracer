#include "hip/hip_runtime.h"
#include "auxiliary.h"
#include "float3_utils.h"
#include "grid.h"
#include "raytrace.h"

#include <hip/hip_runtime.h>
#include <>
#include <functional>

__device__ float3 computeColorFromSH(int idx, int deg, int max_coeffs,
                                     const float3 *means, float3 campos,
                                     const float *shs, bool *clamped) {
  // The implementation is loosely based on code for
  // "Differentiable Point-Based Radiance Fields for
  // Efficient View Synthesis" by Zhang et al. (2022)
  float3 pos_float3 = means[idx];
  float3 pos = make_float3(pos_float3.x, pos_float3.y, pos_float3.z);
  float3 dir = normalize(pos - campos);

  float3 *sh = ((float3 *)shs) + idx * max_coeffs;
  float3 result = SH_C0 * sh[0];

  if (deg > 0) {
    float x = dir.x;
    float y = dir.y;
    float z = dir.z;
    result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

    if (deg > 1) {
      float xx = x * x, yy = y * y, zz = z * z;
      float xy = x * y, yz = y * z, xz = x * z;
      result = result + SH_C2[0] * xy * sh[4] + SH_C2[1] * yz * sh[5] +
               SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
               SH_C2[3] * xz * sh[7] + SH_C2[4] * (xx - yy) * sh[8];

      if (deg > 2) {
        result = result + SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
                 SH_C3[1] * xy * z * sh[10] +
                 SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
                 SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
                 SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
                 SH_C3[5] * z * (xx - yy) * sh[14] +
                 SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
      }
    }
  }
  result = result + make_float3(0.5f, 0.5f, 0.5f);

  // RGB colors are clamped to positive values. If values are
  // clamped, we need to keep track of this for the backward pass.
  clamped[3 * idx + 0] = (result.x < 0);
  clamped[3 * idx + 1] = (result.y < 0);
  clamped[3 * idx + 2] = (result.z < 0);
  return max(result, make_float3(0.0f, 0.0f, 0.0f));
}

__global__ void traceRays(GridCell<64> *grid, float3 const *ellipsoidCenters,
                          float3 const *ellipsoidRadii, float4 const *rotations,
                          float scale_modifier, float3 cam_pos, float3 gridMin,
                          float3 cellSize, int cellsPerAxis, int numEllipsoids,
                          float tan_fovx, float tan_fovy, int width, int height,
                          const float *shs, int sh_deg, int max_coeffs,
                          float *out_color) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  float u = (2.0f * (x + 0.5f) / width - 1.0f) * tan_fovx;
  float v = (1.0f - 2.0f * (y + 0.5f) / height) * tan_fovy;

  float3 rayDir = normalize(make_float3(u, v, 1.0f));

  float3 tMin = (gridMin - cam_pos) / rayDir;
  float3 tMax =
      (gridMin +
       make_float3(cellsPerAxis * cellSize.x, cellsPerAxis * cellSize.y,
                   cellsPerAxis * cellSize.z) -
       cam_pos) /
      rayDir;

  float3 tEnter = min(tMin, tMax);
  float3 tExit = max(tMin, tMax);

  float tStart = max(0.0f, max(tEnter.x, max(tEnter.y, tEnter.z)));
  float tEnd = min(tExit.x, min(tExit.y, tExit.z));

  if (tStart >= tEnd)
    return;

  float3 hitColor = make_float3(0, 0, 0);
  float tMinHit = FLT_MAX;
  int hitEllipsoidIdx = -1;

  int3 step = make_int3(signbit(rayDir.x) ? -1 : 1, signbit(rayDir.y) ? -1 : 1,
                        signbit(rayDir.z) ? -1 : 1);

  int3 gridIndex =
      make_int3((cam_pos.x + tStart * rayDir.x - gridMin.x) / cellSize.x,
                (cam_pos.y + tStart * rayDir.y - gridMin.y) / cellSize.y,
                (cam_pos.z + tStart * rayDir.z - gridMin.z) / cellSize.z);

  float3 nextBoundary =
      make_float3((gridIndex.x + (step.x > 0)) * cellSize.x + gridMin.x,
                  (gridIndex.y + (step.y > 0)) * cellSize.y + gridMin.y,
                  (gridIndex.z + (step.z > 0)) * cellSize.z + gridMin.z);

  float3 tNext = (nextBoundary - cam_pos) / rayDir;
  float3 deltaT =
      make_float3(fabsf(cellSize.x / rayDir.x), fabsf(cellSize.y / rayDir.y),
                  fabsf(cellSize.z / rayDir.z));

  while (gridIndex.x >= 0 && gridIndex.x < cellsPerAxis && gridIndex.y >= 0 &&
         gridIndex.y < cellsPerAxis && gridIndex.z >= 0 &&
         gridIndex.z < cellsPerAxis && tStart < tEnd) {
    int cellIdx =
        gridIndex.x + cellsPerAxis * (gridIndex.y + cellsPerAxis * gridIndex.z);
    GridCell<64> cell = grid[cellIdx];

    for (int j = 0; j < cell.pointCount; j++) {
      int ellipsoidIdx = cell.point_idx[j];
      if (ellipsoidIdx >= numEllipsoids)
        continue;

      float3 center = ellipsoidCenters[ellipsoidIdx];
      float3 radii = ellipsoidRadii[ellipsoidIdx] * scale_modifier;

      float tNear;
      if (rayIntersectsEllipsoid(cam_pos, rayDir, center, radii, tNear) &&
          tNear < tMinHit) {
        tMinHit = tNear;
        hitEllipsoidIdx = ellipsoidIdx;
      }
    }

    if (tNext.x < tNext.y && tNext.x < tNext.z) {
      tStart = tNext.x;
      gridIndex.x += step.x;
      tNext.x += deltaT.x;
    } else if (tNext.y < tNext.z) {
      tStart = tNext.y;
      gridIndex.y += step.y;
      tNext.y += deltaT.y;
    } else {
      tStart = tNext.z;
      gridIndex.z += step.z;
      tNext.z += deltaT.z;
    }
  }

  int pixelIdx = (y * width + x) * 3;
  if (hitEllipsoidIdx >= 0) {
    bool clamped[3];
    float3 campos_glm = make_float3(cam_pos.x, cam_pos.y, cam_pos.z);

    float4 rotation = rotations[hitEllipsoidIdx];
    float4 inverseRotation =
        make_float4(-rotation.x, -rotation.y, -rotation.z, rotation.w);
    float3 rotated_dir = applyRotation(rayDir, inverseRotation);

    // Temporarily modify campos to create an unrotated view direction for
    // computeColorFromSH We'll construct a fake camera position that would
    // result in our rotated direction
    float3 modified_campos = ellipsoidCenters[hitEllipsoidIdx] - rotated_dir;

    float3 color =
        computeColorFromSH(hitEllipsoidIdx, sh_deg, max_coeffs,
                           ellipsoidCenters, modified_campos, shs, clamped);

    out_color[pixelIdx] = color.x;
    out_color[pixelIdx + 1] = color.y;
    out_color[pixelIdx + 2] = color.z;
  } else {
    out_color[pixelIdx] = 0.0f;
    out_color[pixelIdx + 1] = 0.0f;
    out_color[pixelIdx + 2] = 0.0f;
  }
}

int Raytracer::forward(std::function<char *(size_t)> geometryBuffer,
                       std::function<char *(size_t)> binningBuffer,
                       std::function<char *(size_t)> imageBuffer, const int P,
                       int D, int M, const float *background, const int width,
                       int height, const float3 *means3D, const float *shs,
                       const float3 *scales, const float scale_modifier,
                       const float4 *rotations, const float *cov3D_precomp,
                       const float *viewmatrix, const float *projmatrix,
                       const float *cam_pos, const float tan_fovx,
                       float tan_fovy, const bool prefiltered, float *out_color,
                       int *radii, bool debug) {
  accelGrid->build(means3D, scales, P);

  GridCell<64> *d_grid = accelGrid->getDeviceGrid();
  float3 gridMin = accelGrid->getGridMin();
  float3 cellSize = accelGrid->getCellSize();
  int cellsPerAxis = accelGrid->getCellsPerAxis();

  dim3 blockSize(16, 16);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                (height + blockSize.y - 1) / blockSize.y);
  float3 camPos = {cam_pos[0], cam_pos[1], cam_pos[2]};

  traceRays<<<gridSize, blockSize>>>(d_grid, means3D, scales, rotations,
                                     scale_modifier, camPos, gridMin, cellSize,
                                     cellsPerAxis, P, tan_fovx, tan_fovy, width,
                                     height, shs, D, M, out_color);

  return 0;
}
